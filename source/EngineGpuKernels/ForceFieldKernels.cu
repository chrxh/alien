#include "hip/hip_runtime.h"
﻿#include "ForceFieldKernels.cuh"

#include "EngineInterface/SimulationParameters.h"

#include "ConstantMemory.cuh"
#include "ParameterCalculator.cuh"

namespace
{
    __device__ float getHeight(BaseMap const& map, float2 const& pos, int const& index)
    {
        auto dist =
            map.getDistance(pos, float2{cudaSimulationParameters.layerPosition.layerValues[index].x, cudaSimulationParameters.layerPosition.layerValues[index].y});
        if (Orientation_Clockwise == cudaSimulationParameters.layerRadialForceFieldOrientation.layerValues[index]) {
            return sqrtf(dist) * cudaSimulationParameters.layerRadialForceFieldStrength.layerValues[index];
        } else {
            return -sqrtf(dist) * cudaSimulationParameters.layerRadialForceFieldStrength.layerValues[index];
        }
    }

    __device__ __inline__ float2 calcAcceleration(BaseMap const& map, float2 const& pos, int const& index)
    {
        switch (cudaSimulationParameters.layerForceFieldType.layerValues[index].value) {
        case ForceField_Radial: {
            auto baseValue = getHeight(map, pos, index);
            auto downValue = getHeight(map, pos + float2{0, 1}, index);
            auto rightValue = getHeight(map, pos + float2{1, 0}, index);
            float2 result{rightValue - baseValue, downValue - baseValue};
            result = Math::rotateClockwise(
                result, 90.0f + cudaSimulationParameters.layerRadialForceFieldDriftAngle.layerValues[index]);
            return result;
        }
        case ForceField_Central: {
            auto centerDirection = map.getCorrectedDirection(
                float2{cudaSimulationParameters.layerPosition.layerValues[index].x, cudaSimulationParameters.layerPosition.layerValues[index].y} - pos);
            return centerDirection * cudaSimulationParameters.layerCentralForceFieldStrength.layerValues[index]
                / (Math::lengthSquared(centerDirection) + 50.0f);
        }
        case ForceField_Linear: {
            auto centerDirection = Math::unitVectorOfAngle(cudaSimulationParameters.layerLinearForceFieldAngle.layerValues[index]);
            return centerDirection * cudaSimulationParameters.layerLinearForceFieldStrength.layerValues[index];
        }
        default:
            return {0, 0};
        }

    }

}

__global__ void cudaApplyForceFieldSettings(SimulationData data)
{
    float2 accelerations[MAX_LAYERS];

    auto calcResultingAcceleration = [&](float2 const& pos) {
        for (int i = 0; i < cudaSimulationParameters.numLayers; ++i) {
            if (cudaSimulationParameters.layerForceFieldType.layerValues[i].enabled) {
                accelerations[i] = calcAcceleration(data.cellMap, pos, i);
            }
        }
        return ParameterCalculator::calcParameter(float2{0, 0}, accelerations, data, pos);
    };
    {
        auto& cells = data.objects.cellPointers;
        auto partition = calcAllThreadsPartition(cells.getNumEntries());

        for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
            auto& cell = cells.at(index);
            if (cell->barrier) {
                continue;
            }
            cell->shared1 += calcResultingAcceleration(cell->pos);
        }
    }
    {
        auto& particles = data.objects.particlePointers;
        auto partition = calcAllThreadsPartition(particles.getNumEntries());
        for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
            auto& particle = particles.at(index);
            particle->vel += calcResultingAcceleration(particle->pos);
        }
    }
}
