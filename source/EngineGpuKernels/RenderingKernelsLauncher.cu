#include "hip/hip_runtime.h"
﻿#include "RenderingKernelsLauncher.cuh"

#include "RenderingData.cuh"
#include "RenderingKernels.cuh"

void _RenderingKernelsLauncher::drawImage(
    Settings const& settings,
    float2 rectUpperLeft,
    float2 rectLowerRight,
    int2 imageSize,
    float zoom,
    SimulationData data,
    RenderingData renderingData)
{
    uint64_t* targetImage = renderingData.imageData;
    auto const& gpuSettings = settings.gpuSettings;

    KERNEL_CALL(cudaDrawBackground, targetImage, imageSize, data.worldSize, zoom, rectUpperLeft, rectLowerRight);
    KERNEL_CALL_1_1(cudaPrepareFilteringForRendering, data.tempObjects.cellPointers, data.tempObjects.particlePointers);
    KERNEL_CALL(cudaFilterCellsForRendering, data.worldSize, rectUpperLeft, data.objects.cellPointers, data.tempObjects.cellPointers, imageSize, zoom);
    KERNEL_CALL(cudaFilterParticlesForRendering, data.worldSize, rectUpperLeft, data.objects.particlePointers, data.tempObjects.particlePointers, imageSize, zoom);
    KERNEL_CALL(cudaDrawCells, data.timestep, data.worldSize, rectUpperLeft, rectLowerRight, data.tempObjects.cellPointers, targetImage, imageSize, zoom);
    KERNEL_CALL(cudaDrawParticles, data.worldSize, rectUpperLeft, rectLowerRight, data.tempObjects.particlePointers, targetImage, imageSize, zoom);
    KERNEL_CALL_1_1(cudaDrawRadiationSources, targetImage, rectUpperLeft, data.worldSize, imageSize, zoom);

    if (settings.simulationParameters.features.cellGlow) {
        cudaDrawCellGlow<<<512, 128>>>(data.worldSize, rectUpperLeft, data.tempObjects.cellPointers, targetImage, imageSize, zoom);
    }

    if (settings.simulationParameters.borderlessRendering) {
        KERNEL_CALL(cudaDrawRepetition, data.worldSize, imageSize, rectUpperLeft, rectLowerRight, targetImage, zoom);
    }
}
