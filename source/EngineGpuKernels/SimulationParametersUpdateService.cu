#include "hip/hip_runtime.h"
#include "SimulationParametersUpdateService.cuh"

#include <vector>

#include "EngineInterface/SimulationParameters.h"
#include "EngineInterface/SpaceCalculator.h"

#include "Base.cuh"
#include "SimulationData.cuh"
#include "MaxAgeBalancer.cuh"

SimulationParameters SimulationParametersUpdateService::integrateChanges(
    SimulationParameters const& currentParameters,
    SimulationParameters const& changedParameters,
    SimulationParametersUpdateConfig const& updateConfig) const
{
    auto result = changedParameters;

    if (updateConfig == SimulationParametersUpdateConfig::AllExceptChangingPositions) {
        auto numSpots = std::min(currentParameters.numZones.value, changedParameters.numZones.value);
        for (int i = 0; i < numSpots; ++i) {
            if (currentParameters.zoneVelocity.zoneValues[i].x != 0) {
                result.zonePosition.zoneValues[i].x = currentParameters.zonePosition.zoneValues[i].x;
            }
            if (currentParameters.zoneVelocity.zoneValues[i].y != 0) {
                result.zonePosition.zoneValues[i].y = currentParameters.zonePosition.zoneValues[i].y;
            }
        }

        auto numRadiationSources = std::min(currentParameters.numZones.value, changedParameters.numZones.value);
        for (int i = 0; i < numRadiationSources; ++i) {
            if (currentParameters.radiationSource[i].velX != 0) {
                result.radiationSource[i].posX = currentParameters.radiationSource[i].posX;
            }
            if (currentParameters.radiationSource[i].velY != 0) {
                result.radiationSource[i].posY = currentParameters.radiationSource[i].posY;
            }
        }
    }
    return result;
}

bool SimulationParametersUpdateService::updateSimulationParametersAfterTimestep(
    SettingsForSimulation& settings,
    MaxAgeBalancer const& maxAgeBalancer,
    SimulationData const& simulationData,
    StatisticsRawData const& statistics)
{
    auto result = false;

    auto const& worldSizeX = settings.worldSizeX;
    auto const& worldSizeY = settings.worldSizeY;
    SpaceCalculator space({worldSizeX, worldSizeY});
    for (int i = 0; i < settings.simulationParameters.numZones.value; ++i) {
        auto& source = settings.simulationParameters.radiationSource[i];
        if (abs(source.velX) > NEAR_ZERO) {
            source.posX += source.velX * settings.simulationParameters.timestepSize.value;
            result = true;
        }
        if (abs(source.velY) > NEAR_ZERO) {
            source.posY += source.velY * settings.simulationParameters.timestepSize.value;
            result = true;
        }
        auto correctedPosition = space.getCorrectedPosition({source.posX, source.posY});
        source.posX = correctedPosition.x;
        source.posY = correctedPosition.y;
    }
    for (int i = 0; i < settings.simulationParameters.numZones.value; ++i) {
        auto& zonePosition = settings.simulationParameters.zonePosition.zoneValues[i];
        auto& zoneVelocity = settings.simulationParameters.zoneVelocity.zoneValues[i];
        if (abs(zoneVelocity.x) > NEAR_ZERO) {
            zonePosition.x += zoneVelocity.x * settings.simulationParameters.timestepSize.value;
            result = true;
        }
        if (abs(zoneVelocity.y) > NEAR_ZERO) {
            zonePosition.y += zoneVelocity.y * settings.simulationParameters.timestepSize.value;
            result = true;
        }
        zonePosition = space.getCorrectedPosition(zonePosition);
    }

    auto externalEnergyPresent = settings.simulationParameters.externalEnergy.value > 0;
    for (int i = 0; i < MAX_COLORS; ++i) {
        externalEnergyPresent |= settings.simulationParameters.externalEnergyBackflowFactor.value[i] > 0;
    }
    externalEnergyPresent &= settings.simulationParameters.externalEnergyControlToggle.value;
    if (externalEnergyPresent) {
        double temp;
        CHECK_FOR_CUDA_ERROR(hipMemcpy(&temp, simulationData.externalEnergy, sizeof(double), hipMemcpyDeviceToHost));
        settings.simulationParameters.externalEnergy.value = toFloat(temp);
        result = true;
    }

    result |= maxAgeBalancer->balance(settings.simulationParameters, statistics, simulationData.timestep);

    return result;
}
