#include "hip/hip_runtime.h"
#include "SimulationParametersUpdateService.cuh"

#include <vector>

#include "EngineInterface/SimulationParameters.h"
#include "EngineInterface/SpaceCalculator.h"

#include "Base.cuh"
#include "SimulationData.cuh"
#include "MaxAgeBalancer.cuh"

SimulationParameters SimulationParametersUpdateService::integrateChanges(
    SimulationParameters const& currentParameters,
    SimulationParameters const& changedParameters,
    SimulationParametersUpdateConfig const& updateConfig) const
{
    auto result = changedParameters;

    if (updateConfig == SimulationParametersUpdateConfig::AllExceptChangingPositions) {
        auto numSpots = std::min(currentParameters.numSpots, changedParameters.numSpots);
        for (int i = 0; i < numSpots; ++i) {
            if (currentParameters.spots[i].velX != 0) {
                result.spots[i].posX = currentParameters.spots[i].posX;
            }
            if (currentParameters.spots[i].velY != 0) {
                result.spots[i].posY = currentParameters.spots[i].posY;
            }
        }

        auto numRadiationSources = std::min(currentParameters.numRadiationSources, changedParameters.numRadiationSources);
        for (int i = 0; i < numRadiationSources; ++i) {
            if (currentParameters.radiationSources[i].velX != 0) {
                result.radiationSources[i].posX = currentParameters.radiationSources[i].posX;
            }
            if (currentParameters.radiationSources[i].velY != 0) {
                result.radiationSources[i].posY = currentParameters.radiationSources[i].posY;
            }
        }
    }
    return result;
}

bool SimulationParametersUpdateService::updateSimulationParametersAfterTimestep(
    Settings& settings,
    MaxAgeBalancer const& maxAgeBalancer,
    SimulationData const& simulationData,
    RawStatisticsData const& statistics)
{
    auto result = false;

    auto const& worldSizeX = settings.generalSettings.worldSizeX;
    auto const& worldSizeY = settings.generalSettings.worldSizeY;
    SpaceCalculator space({worldSizeX, worldSizeY});
    for (int i = 0; i < settings.simulationParameters.numRadiationSources; ++i) {
        auto& source = settings.simulationParameters.radiationSources[i];
        if (abs(source.velX) > NEAR_ZERO) {
            source.posX += source.velX * settings.simulationParameters.timestepSize;
            result = true;
        }
        if (abs(source.velY) > NEAR_ZERO) {
            source.posY += source.velY * settings.simulationParameters.timestepSize;
            result = true;
        }
        auto correctedPosition = space.getCorrectedPosition({source.posX, source.posY});
        source.posX = correctedPosition.x;
        source.posY = correctedPosition.y;
    }
    for (int i = 0; i < settings.simulationParameters.numSpots; ++i) {
        auto& spot = settings.simulationParameters.spots[i];
        if (abs(spot.velX) > NEAR_ZERO) {
            spot.posX += spot.velX * settings.simulationParameters.timestepSize;
            result = true;
        }
        if (abs(spot.velY) > NEAR_ZERO) {
            spot.posY += spot.velY * settings.simulationParameters.timestepSize;
            result = true;
        }
        auto correctedPosition = space.getCorrectedPosition({spot.posX, spot.posY});
        spot.posX = correctedPosition.x;
        spot.posY = correctedPosition.y;
    }

    auto externalEnergyPresent = settings.simulationParameters.externalEnergy > 0;
    for (int i = 0; i < MAX_COLORS; ++i) {
        externalEnergyPresent |= settings.simulationParameters.externalEnergyBackflowFactor[i] > 0;
    }
    externalEnergyPresent &= settings.simulationParameters.features.externalEnergyControl;
    if (externalEnergyPresent) {
        double temp;
        CHECK_FOR_CUDA_ERROR(hipMemcpy(&temp, simulationData.externalEnergy, sizeof(double), hipMemcpyDeviceToHost));
        settings.simulationParameters.externalEnergy = toFloat(temp);
        result = true;
    }

    result |= maxAgeBalancer->balance(settings.simulationParameters, statistics, simulationData.timestep);

    return result;
}
