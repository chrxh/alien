#include "hip/hip_runtime.h"
#include "SimulationParametersUpdateService.cuh"

#include <vector>

#include "EngineInterface/SimulationParameters.h"
#include "EngineInterface/SpaceCalculator.h"

#include "Base.cuh"
#include "SimulationData.cuh"
#include "MaxAgeBalancer.cuh"

SimulationParameters SimulationParametersUpdateService::integrateChanges(
    SimulationParameters const& currentParameters,
    SimulationParameters const& changedParameters,
    SimulationParametersUpdateConfig const& updateConfig) const
{
    auto result = changedParameters;

    if (updateConfig == SimulationParametersUpdateConfig::AllExceptChangingPositions) {
        auto numSpots = std::min(currentParameters.numZones, changedParameters.numZones);
        for (int i = 0; i < numSpots; ++i) {
            if (currentParameters.zone[i].velX != 0) {
                result.zone[i].posX = currentParameters.zone[i].posX;
            }
            if (currentParameters.zone[i].velY != 0) {
                result.zone[i].posY = currentParameters.zone[i].posY;
            }
        }

        auto numRadiationSources = std::min(currentParameters.numRadiationSources, changedParameters.numRadiationSources);
        for (int i = 0; i < numRadiationSources; ++i) {
            if (currentParameters.radiationSource[i].velX != 0) {
                result.radiationSource[i].posX = currentParameters.radiationSource[i].posX;
            }
            if (currentParameters.radiationSource[i].velY != 0) {
                result.radiationSource[i].posY = currentParameters.radiationSource[i].posY;
            }
        }
    }
    return result;
}

bool SimulationParametersUpdateService::updateSimulationParametersAfterTimestep(
    Settings& settings,
    MaxAgeBalancer const& maxAgeBalancer,
    SimulationData const& simulationData,
    RawStatisticsData const& statistics)
{
    auto result = false;

    auto const& worldSizeX = settings.generalSettings.worldSizeX;
    auto const& worldSizeY = settings.generalSettings.worldSizeY;
    SpaceCalculator space({worldSizeX, worldSizeY});
    for (int i = 0; i < settings.simulationParameters.numRadiationSources; ++i) {
        auto& source = settings.simulationParameters.radiationSource[i];
        if (abs(source.velX) > NEAR_ZERO) {
            source.posX += source.velX * settings.simulationParameters.timestepSize;
            result = true;
        }
        if (abs(source.velY) > NEAR_ZERO) {
            source.posY += source.velY * settings.simulationParameters.timestepSize;
            result = true;
        }
        auto correctedPosition = space.getCorrectedPosition({source.posX, source.posY});
        source.posX = correctedPosition.x;
        source.posY = correctedPosition.y;
    }
    for (int i = 0; i < settings.simulationParameters.numZones; ++i) {
        auto& spot = settings.simulationParameters.zone[i];
        if (abs(spot.velX) > NEAR_ZERO) {
            spot.posX += spot.velX * settings.simulationParameters.timestepSize;
            result = true;
        }
        if (abs(spot.velY) > NEAR_ZERO) {
            spot.posY += spot.velY * settings.simulationParameters.timestepSize;
            result = true;
        }
        auto correctedPosition = space.getCorrectedPosition({spot.posX, spot.posY});
        spot.posX = correctedPosition.x;
        spot.posY = correctedPosition.y;
    }

    auto externalEnergyPresent = settings.simulationParameters.externalEnergy > 0;
    for (int i = 0; i < MAX_COLORS; ++i) {
        externalEnergyPresent |= settings.simulationParameters.externalEnergyBackflowFactor[i] > 0;
    }
    externalEnergyPresent &= settings.simulationParameters.features.externalEnergyControl;
    if (externalEnergyPresent) {
        double temp;
        CHECK_FOR_CUDA_ERROR(hipMemcpy(&temp, simulationData.externalEnergy, sizeof(double), hipMemcpyDeviceToHost));
        settings.simulationParameters.externalEnergy = toFloat(temp);
        result = true;
    }

    result |= maxAgeBalancer->balance(settings.simulationParameters, statistics, simulationData.timestep);

    return result;
}
