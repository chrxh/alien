#include "hip/hip_runtime.h"
﻿#include "FlowFieldKernels.cuh"

#include "EngineInterface/SimulationParameters.h"

#include "ConstantMemory.cuh"
#include "SpotCalculator.cuh"

namespace
{
    __device__ float getHeight(BaseMap const& map, float2 const& pos, SimulationParametersZone const& spot)
    {
        auto dist = map.getDistance(pos, float2{spot.posX, spot.posY});
        if (Orientation_Clockwise == spot.flowData.radialFlow.orientation) {
            return sqrtf(dist) * spot.flowData.radialFlow.strength;
        } else {
            return -sqrtf(dist) * spot.flowData.radialFlow.strength;
        }
    }

    __device__ __inline__ float2 calcAcceleration(BaseMap const& map, float2 const& pos, int const& spotIndex)
    {
        auto const& spot = cudaSimulationParameters.zone[spotIndex];
        switch (spot.flowType) {
        case FlowType_Radial: {
            auto baseValue = getHeight(map, pos, spot);
            auto downValue = getHeight(map, pos + float2{0, 1}, spot);
            auto rightValue = getHeight(map, pos + float2{1, 0}, spot);
            float2 result{rightValue - baseValue, downValue - baseValue};
            result = Math::rotateClockwise(result, 90.0f + spot.flowData.radialFlow.driftAngle);
            return result;
        }
        case FlowType_Central: {
            auto centerDirection = map.getCorrectedDirection(float2{spot.posX, spot.posY} - pos);
            return centerDirection * spot.flowData.centralFlow.strength / (Math::lengthSquared(centerDirection) + 50.0f);
        }
        case FlowType_Linear: {
            auto centerDirection = Math::unitVectorOfAngle(spot.flowData.linearFlow.angle);
            return centerDirection * spot.flowData.linearFlow.strength;
        }
        default:
            return {0, 0};
        }

    }

}

__global__ void cudaApplyFlowFieldSettings(SimulationData data)
{
    float2 accelerations[MAX_ZONES];
    {
        auto& cells = data.objects.cellPointers;
        auto partition = calcAllThreadsPartition(cells.getNumEntries());

        for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
            auto& cell = cells.at(index);
            if (cell->barrier) {
                continue;
            }
            int numFlowFields = 0;
            for (int i = 0; i < cudaSimulationParameters.numZones; ++i) {

                if (cudaSimulationParameters.zone[i].flowType != FlowType_None) {
                    accelerations[numFlowFields] = calcAcceleration(data.cellMap, cell->pos, i);
                    ++numFlowFields;
                }
            }
            auto resultingAcceleration = SpotCalculator::calcResultingFlowField(data.cellMap, cell->pos, float2{0, 0}, accelerations);
            cell->shared1 += resultingAcceleration;
        }
    }
    {
        auto& particles = data.objects.particlePointers;
        auto partition = calcAllThreadsPartition(particles.getNumEntries());
        for (int index = partition.startIndex; index <= partition.endIndex; ++index) {
            auto& particle = particles.at(index);
            int numFlowFields = 0;
            for (int i = 0; i < cudaSimulationParameters.numZones; ++i) {

                if (cudaSimulationParameters.zone[i].flowType != FlowType_None) {
                    accelerations[numFlowFields] = calcAcceleration(data.cellMap, particle->absPos, i);
                    ++numFlowFields;
                }
            }
            auto resultingAcceleration = SpotCalculator::calcResultingFlowField(data.cellMap, particle->absPos, float2{0, 0}, accelerations);
            particle->vel += resultingAcceleration;
        }
    }
}
