﻿#include "GarbageCollectorKernelsService.cuh"

_GarbageCollectorKernelsService::_GarbageCollectorKernelsService()
{
    CudaMemoryManager::getInstance().acquireMemory<bool>(1, _cudaBool);
}

_GarbageCollectorKernelsService::~_GarbageCollectorKernelsService()
{
    CudaMemoryManager::getInstance().freeMemory(_cudaBool);
}

void _GarbageCollectorKernelsService::cleanupAfterTimestep(GpuSettings const& gpuSettings, SimulationData const& data)
{
    KERNEL_CALL(cudaCleanupCellMap, data);
    KERNEL_CALL(cudaCleanupParticleMap, data);

    KERNEL_CALL_1_1(cudaPreparePointerArraysForCleanup, data);
    KERNEL_CALL(cudaCleanupPointerArray<Particle*>, data.objects.particles, data.tempObjects.particles);
    KERNEL_CALL(cudaCleanupPointerArray<Cell*>, data.objects.cells, data.tempObjects.cells);
    KERNEL_CALL_1_1(cudaSwapPointerArrays, data);

    KERNEL_CALL_1_1(cudaCheckIfCleanupIsNecessary, data, _cudaBool);
    hipDeviceSynchronize();
    if (copyToHost(_cudaBool)) {
        KERNEL_CALL_1_1(cudaPrepareHeapForCleanup, data);
        KERNEL_CALL(cudaCleanupParticles, data.objects.particles, data.tempObjects.heap);
        KERNEL_CALL(cudaCleanupGenomesStep1, data.objects.cells);
        KERNEL_CALL(cudaCleanupGenomesStep2, data.objects.cells, data.tempObjects.heap);
        KERNEL_CALL(cudaCleanupGenomesStep3, data.objects.cells, data.tempObjects.heap);
        KERNEL_CALL(cudaCleanupCellsStep1, data.objects.cells, data.tempObjects.heap);
        KERNEL_CALL(cudaCleanupCellsStep2, data.objects.cells, data.tempObjects.heap);
        KERNEL_CALL(cudaCleanupDependentCellData, data.objects.cells, data.tempObjects.heap);
        KERNEL_CALL_1_1(cudaSwapHeaps, data);
    }
}

void _GarbageCollectorKernelsService::cleanupAfterDataManipulation(GpuSettings const& gpuSettings, SimulationData const& data)
{
    KERNEL_CALL_1_1(cudaPreparePointerArraysForCleanup, data);
    KERNEL_CALL(cudaCleanupPointerArray<Particle*>, data.objects.particles, data.tempObjects.particles);
    KERNEL_CALL(cudaCleanupPointerArray<Cell*>, data.objects.cells, data.tempObjects.cells);
    KERNEL_CALL_1_1(cudaSwapPointerArrays, data);

    KERNEL_CALL_1_1(cudaPrepareHeapForCleanup, data);
    KERNEL_CALL(cudaCleanupParticles, data.objects.particles, data.tempObjects.heap);
    KERNEL_CALL(cudaCleanupGenomesStep1, data.objects.cells);
    KERNEL_CALL(cudaCleanupGenomesStep2, data.objects.cells, data.tempObjects.heap);
    KERNEL_CALL(cudaCleanupGenomesStep3, data.objects.cells, data.tempObjects.heap);
    KERNEL_CALL(cudaCleanupCellsStep1, data.objects.cells, data.tempObjects.heap);
    KERNEL_CALL(cudaCleanupCellsStep2, data.objects.cells, data.tempObjects.heap);
    KERNEL_CALL(cudaCleanupDependentCellData, data.objects.cells, data.tempObjects.heap);
    KERNEL_CALL_1_1(cudaSwapHeaps, data);
}

void _GarbageCollectorKernelsService::copyArrays(GpuSettings const& gpuSettings, SimulationData const& data)
{
    KERNEL_CALL_1_1(cudaPreparePointerArraysForCleanup, data);
    KERNEL_CALL(cudaCleanupPointerArray<Particle*>, data.objects.particles, data.tempObjects.particles);
    KERNEL_CALL(cudaCleanupPointerArray<Cell*>, data.objects.cells, data.tempObjects.cells);

    KERNEL_CALL_1_1(cudaPrepareHeapForCleanup, data);
    KERNEL_CALL(cudaCleanupParticles, data.tempObjects.particles, data.tempObjects.heap);
    KERNEL_CALL(cudaCleanupGenomesStep1, data.objects.cells);
    KERNEL_CALL(cudaCleanupGenomesStep2, data.objects.cells, data.tempObjects.heap);
    KERNEL_CALL(cudaCleanupGenomesStep3, data.objects.cells, data.tempObjects.heap);
    KERNEL_CALL(cudaCleanupCellsStep1, data.tempObjects.cells, data.tempObjects.heap);
    KERNEL_CALL(cudaCleanupCellsStep2, data.tempObjects.cells, data.tempObjects.heap);
    KERNEL_CALL(cudaCleanupDependentCellData, data.tempObjects.cells, data.tempObjects.heap);
}

void _GarbageCollectorKernelsService::swapArrays(GpuSettings const& gpuSettings, SimulationData const& data)
{
    KERNEL_CALL_1_1(cudaSwapPointerArrays, data);
    KERNEL_CALL_1_1(cudaSwapHeaps, data);
}
